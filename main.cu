#include <iostream>
#include "data_transfers/pinning.cuh"
#include "data_transfers/staged_copy_execute.cuh"

int main() {
    hipDeviceProp_t prop;
    check(hipGetDeviceProperties(&prop, 0));
    std::cout << "Device used to perform measurements: " << prop.name << std::endl;

    // page-locked memory transfer measurement
    int numElements = 4 * 1024 * 1024;
    measurePinning(numElements);

    if (prop.deviceOverlap) // see if concurrent copy & execute is supported by the GPU
    {
        stagedCopyExecute(512, 16);
    }

    return 0;
}
