#include <stdio.h>
#include "pinning.cuh"

void copyProfiler(float *h_a, float *h_b, float *d, unsigned int n) {
    unsigned int bytes = n * sizeof(float);

    hipEvent_t start, stop;

    check(hipEventCreate(&start));
    check(hipEventCreate(&stop));

    check(hipEventRecord(start, 0));
    check(hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice));
    check(hipEventRecord(stop, 0));
    check(hipEventSynchronize(stop));

    float time;
    check(hipEventElapsedTime(&time, start, stop));
    std::cout << "  Host to Device Bandwidth in GB/s: " << bytes * 1e-6 / time << std::endl;

    check(hipEventRecord(start, 0));
    check(hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost));
    check(hipEventRecord(stop, 0));
    check(hipEventSynchronize(stop));

    check(hipEventElapsedTime(&time, start, stop));
    std::cout << "  Device to Host Bandwidth in GB/s: " << bytes * 1e-6 / time << std::endl;

    check(hipEventDestroy(start));
    check(hipEventDestroy(stop));
}

void measurePinning(int numElements) {
    const unsigned int bytes = numElements * sizeof(float);
    float *h_aPageable, *h_bPageable, *h_aPinned, *h_bPinned, *d_p;

    std::cout << "Pageable vs Pinned memory transfer comparison\n Transfer size in MB: " << bytes / (1024 * 1024)
              << std::endl;

    // allocate pageable and pinned host memory and device memory
    h_aPageable = (float *) malloc(bytes);
    h_bPageable = (float *) malloc(bytes);
    check(hipHostMalloc((void **) &h_aPinned, bytes));
    check(hipHostMalloc((void **) &h_bPinned, bytes));
    check(hipMalloc((void **) &d_p, bytes));

    for (int i = 0; i < numElements; ++i) {
        h_aPageable[i] = i;
        h_aPinned[i] = i;
    }

    memset(h_bPageable, 0, bytes);
    memset(h_bPinned, 0, bytes);

    std::cout << "Pageable transfer:" << std::endl;
    copyProfiler(h_aPageable, h_bPageable, d_p, numElements);

    std::cout << "Pinned transfer:" << std::endl;
    copyProfiler(h_aPinned, h_bPinned, d_p, numElements);

    hipFree(d_p);
    hipHostFree(h_aPinned);
    hipHostFree(h_bPinned);
    free(h_aPageable);
    free(h_bPageable);
}