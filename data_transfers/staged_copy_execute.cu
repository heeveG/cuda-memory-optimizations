#include "hip/hip_runtime.h"
//
// Created by heeve on 27.05.22.
//

#include "staged_copy_execute.cuh"
#include "../include/util.h"

__global__ void kernel(float *a, int streamOffset) {
    int i = threadIdx.x + blockIdx.x * blockDim.x + streamOffset;

    auto iFloat = (float) i;
    float sin = sinf(iFloat);
    float cos = cosf(iFloat);

    a[i] += sqrtf(sin * sin + cos * cos);
}

void stagedCopyExecute(int blockSize, int numStreams) {

    int numElements = blockSize * numStreams * 8 * 1024;
    int bytes = numElements * sizeof(float);

    int streamSize = numElements / numStreams;
    int streamBytes = streamSize * sizeof(float);

    float *h_a, *d_a;

    check(hipHostMalloc((void **) &h_a, bytes));
    check(hipMalloc((void **) &d_a, bytes));

    hipStream_t streams[numStreams];

    for (auto &stream : streams) check(hipStreamCreate(&stream));

    // sequential version
    memset(h_a, 0, bytes);
    float timeSeq = cudaEventProfile([&]() {
        check(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
        kernel<<<numElements / blockSize, blockSize>>>(d_a, 0);
        check(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));
    });

    std::cout <<"Time for sequential copy-execute in ms " <<  timeSeq << std::endl;

    // async version
    memset(h_a, 0, bytes);
    float timeAsync = cudaEventProfile([&]() {
        for (int i = 0; i < numStreams; ++i) {
            int offset = i * streamSize;
            check(hipMemcpyAsync(&d_a[offset], &h_a[offset], streamBytes, hipMemcpyHostToDevice,
                                  streams[i]));
        }
        for (int i = 0; i < numStreams; ++i) {
            int offset = i * streamSize;
            kernel<<<streamSize / blockSize, blockSize, 0, streams[i]>>>(d_a, offset);
        }
        for (int i = 0; i < numStreams; ++i) {
            int offset = i * streamSize;
            check(hipMemcpyAsync(&h_a[offset], &d_a[offset], streamBytes, hipMemcpyDeviceToHost,
                                  streams[i]));
        }
    });

    std::cout << "Time for asynchronous copy-execute in ms: " << timeAsync << std::endl;

    for (auto& stream : streams) check(hipStreamDestroy(stream));

    hipFree(d_a);
    hipHostFree(h_a);
}