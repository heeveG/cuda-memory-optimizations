//
// Created by heeve on 27.05.22.
//

#include "batching.cuh"
#include <functional>

void copyProfiler(const std::function<void()> &copyLambdaH2D, const std::function<void()> &copyLambdaD2H) {
    hipEvent_t start, stop;

    check(hipEventCreate(&start));
    check(hipEventCreate(&stop));

    check(hipEventRecord(start, 0));
    copyLambdaH2D();
    check(hipEventRecord(stop, 0));
    check(hipEventSynchronize(stop));

    float time;
    check(hipEventElapsedTime(&time, start, stop));
    std::cout << "  Host to Device Time to copy: " << time << std::endl;

    check(hipEventRecord(start, 0));
    copyLambdaD2H();
    check(hipEventRecord(stop, 0));
    check(hipEventSynchronize(stop));

    check(hipEventElapsedTime(&time, start, stop));
    std::cout << "  Device to Host Time to copy: " << time << std::endl;

    check(hipEventDestroy(start));
    check(hipEventDestroy(stop));
}

void measureBatching() {
    int numElements = 1024 * 1024;
    const unsigned int bytes = numElements * sizeof(float);
    float *h_a, *h_b, *h_c, *h_d, *h_batched, *d_a, *d_b, *d_c, *d_d, *d_batched;

    h_a = (float *) malloc(bytes);
    h_b = (float *) malloc(bytes);
    h_c = (float *) malloc(bytes);
    h_d = (float *) malloc(bytes);
    check(hipMalloc((void **) &d_a, bytes));
    check(hipMalloc((void **) &d_b, bytes));
    check(hipMalloc((void **) &d_c, bytes));
    check(hipMalloc((void **) &d_d, bytes));


    for (int i = 0; i < numElements; ++i) h_a[i] = i;
    memcpy(h_b, h_a, bytes);
    memcpy(h_c, h_a, bytes);
    memcpy(h_d, h_a, bytes);

    // measure without batching
    copyProfiler([&]() {
                     check(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
                     check(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));
                     check(hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice));
                     check(hipMemcpy(d_d, h_d, bytes, hipMemcpyHostToDevice));
                 },
                 [&]() {
                     check(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));
                     check(hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost));
                     check(hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost));
                     check(hipMemcpy(h_d, d_d, bytes, hipMemcpyDeviceToHost));
                 }
    );

    check(hipHostMalloc((void **) &h_batched, 4 * bytes));
    check(hipMalloc((void **) &d_batched, 4 * bytes));

    memcpy(h_batched + 0 * numElements, h_a, bytes);
    memcpy(h_batched + 1 * numElements, h_b, bytes);
    memcpy(h_batched + 2 * numElements, h_c, bytes);
    memcpy(h_batched + 3 * numElements, h_d, bytes);

    // measure with batching
    copyProfiler([&]() {
                     check(hipMemcpy(d_batched, h_batched, 4 * bytes, hipMemcpyHostToDevice));
                 },
                 [&]() {
                     check(hipMemcpy(h_batched, d_batched, 4 * bytes, hipMemcpyDeviceToHost));
                 }
    );

    free(h_a);
    free(h_b);
    free(h_c);
    free(h_d);
    hipHostFree(h_batched);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipFree(d_batched);
}